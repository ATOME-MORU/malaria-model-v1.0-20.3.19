#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <iostream>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <vector>

#include "util/randomness.h"

namespace util {


#ifdef __NVCC__
int cuda_device = 0;
#endif

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return;}} while(0)
    // return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return;}} while(0)
    // return EXIT_FAILURE;}} while(0)

__global__ void filter_ones(float* number_array, int array_length) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < array_length) {
        if (number_array[index] >= 1) {
            number_array[index] -= (1.0-kEPS);
        }
    }
}


void set_random_numbers_uniform_curand(float* result_array, int array_length){
    static hiprandGenerator_t gen;
    static bool seeded(false);

    // CUDA_CALL(hipSetDevice(1));

    static float* dev_random_number_array;
    static int dev_random_number_array_size(1);

    const int kThreads_per_block = 1024;

    CUDA_CALL(
        hipSetDevice(cuda_device)
    );

    if (!seeded) {
        // CURAND_CALL(hiprandCreateGenerator(&gen,
        //  HIPRAND_RNG_PSEUDO_DEFAULT));

            // CURAND_CALL(hiprandCreateGenerator(&gen, 
            //     HIPRAND_RNG_PSEUDO_DEFAULT));
        CURAND_CALL(
            hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT)
        );
        // CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)));
        CURAND_CALL(
            hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL))
        );
        seeded = true;

        CUDA_CALL(
            hipMalloc((void**) &dev_random_number_array, dev_random_number_array_size * sizeof(float))
        );
    }


    if (array_length > dev_random_number_array_size) {
        CUDA_CALL(
            hipFree(dev_random_number_array)
        );
        dev_random_number_array_size = array_length;
        CUDA_CALL(
            hipMalloc((void**) &dev_random_number_array, dev_random_number_array_size * sizeof(float))
        );
    }


    // std::cout << "GPU ramdom\n";
    // CUDA_CALL(hipMalloc((void**) &dev_random_number_array, array_length * sizeof(float)));

    // CURAND_CALL(hiprandGenerateUniform(gen, dev_random_number_array, array_length));

    // CUDA_CALL(hipMemcpy(random_numbers, dev_random_number_array, array_length * sizeof(float)), hipMemcpyDeviceToHost);

    // CUDA_CALL(hipFree(dev_random_number_array));

    CURAND_CALL(
        hiprandGenerateUniform(gen, dev_random_number_array, array_length)
    );

    filter_ones<<<(array_length+kThreads_per_block-1)/kThreads_per_block,kThreads_per_block>>>(dev_random_number_array, array_length);

    CUDA_CALL(
        hipMemcpy(result_array, dev_random_number_array, array_length * sizeof(float), hipMemcpyDeviceToHost)
    );
    // hipFree(dev_random_number_array);

}

}