#include "hip/hip_runtime.h"
#include <iostream>

#include "village/mosquito_ode.cuh"

namespace village {

MosquitoEckhoff_multi::Fx::Fx(
        MosquitoEckhoff_multi::value_t larva_to_immature,
        MosquitoEckhoff_multi::value_t infected_to_infectious,
        MosquitoEckhoff_multi::value_t eggs_to_larva
    ) : 
        kLarva_to_immature(larva_to_immature),
        kInfected_to_infectious(infected_to_infectious),
        kEggs_to_larva(eggs_to_larva) {

}

MosquitoEckhoff_multi::MosquitoEckhoff_multi(
        std::vector<float>& host_max_larval_capacity_list
    ) : 
        kNum_mosqs(host_max_larval_capacity_list.size()),
        kMax_larval_capacity_list(host_max_larval_capacity_list),
        population(9*kNum_mosqs) {

    thrust::fill(population.begin(), population.end(), 0.0);
    thrust::fill(
        population.begin() + 2 * kNum_mosqs,
        population.begin() + 3 * kNum_mosqs,
        200.0
    );

    this->reset_time();
}

void MosquitoEckhoff_multi::reset_time() {
    this->current_time = 0.0;
}

void MosquitoEckhoff_multi::pre_integration_update(
        std::vector<float> host_infection_prob_list
    ) {

    assert(host_infection_prob_list.size() == this->kNum_mosqs);
    this->infection_prob_list = host_infection_prob_list;

    this->current_temperature =
        273.15 + 25.0 + 0.8 * 5.0 * cos(
                                (this->current_time-60)
                                / this->kNum_days_per_year
                                * 2.0 * this->kPi
                            );
    this->current_larva_to_immature =
        this->kAquatic_arrhenius_1
        * exp(-this->kAquatic_arrhenius_2 / this->current_temperature);
    this->current_infected_to_infectious = 
        this->kInfected_arrhenius_1
        * exp(-this->kInfected_arrhenius_2 / this->current_temperature);
    this->current_eggs_to_larva = 
        this->kEgg_arrhenius1
        * exp(-this->kEgg_arrhenius2 / this->current_temperature);


    this->current_time += this->kTime_length_per_integration;
}

void MosquitoEckhoff_multi::operator()(state_t& x, state_t& dxdt, value_t t) const {
    (void) t;
    thrust::for_each(
        thrust::make_zip_iterator( thrust::make_tuple(
            this->kMax_larval_capacity_list.begin(),
            this->infection_prob_list.begin(),
            thrust::make_zip_iterator( thrust::make_tuple(
                x.begin(),                  //[0]
                x.begin() + 1 * kNum_mosqs, //[1]
                x.begin() + 2 * kNum_mosqs, //[2]
                x.begin() + 3 * kNum_mosqs, //[3]
                x.begin() + 4 * kNum_mosqs, //[4]
                x.begin() + 5 * kNum_mosqs, //[5]
                x.begin() + 6 * kNum_mosqs, //[6]
                x.begin() + 7 * kNum_mosqs, //[7]
                x.begin() + 8 * kNum_mosqs  //[8]
            ) ),
            thrust::make_zip_iterator( thrust::make_tuple(
                dxdt.begin(),                  //[0]
                dxdt.begin() + 1 * kNum_mosqs, //[1]
                dxdt.begin() + 2 * kNum_mosqs, //[2]
                dxdt.begin() + 3 * kNum_mosqs, //[3]
                dxdt.begin() + 4 * kNum_mosqs, //[4]
                dxdt.begin() + 5 * kNum_mosqs, //[5]
                dxdt.begin() + 6 * kNum_mosqs, //[6]
                dxdt.begin() + 7 * kNum_mosqs, //[7]
                dxdt.begin() + 8 * kNum_mosqs  //[8]
            ) )
        ) ),
        thrust::make_zip_iterator( thrust::make_tuple(
            this->kMax_larval_capacity_list.end(),
            this->infection_prob_list.end(),
            thrust::make_zip_iterator( thrust::make_tuple(
                x.begin() + 1 * kNum_mosqs, //[0]
                x.begin() + 2 * kNum_mosqs, //[1]
                x.begin() + 3 * kNum_mosqs, //[2]
                x.begin() + 4 * kNum_mosqs, //[3]
                x.begin() + 5 * kNum_mosqs, //[4]
                x.begin() + 6 * kNum_mosqs, //[5]
                x.begin() + 7 * kNum_mosqs, //[6]
                x.begin() + 8 * kNum_mosqs, //[7]
                x.begin() + 9 * kNum_mosqs //[8]
            ) ),
            thrust::make_zip_iterator( thrust::make_tuple(
                dxdt.begin() + 1 * kNum_mosqs, //[0]
                dxdt.begin() + 2 * kNum_mosqs, //[1]
                dxdt.begin() + 3 * kNum_mosqs, //[2]
                dxdt.begin() + 4 * kNum_mosqs, //[3]
                dxdt.begin() + 5 * kNum_mosqs, //[4]
                dxdt.begin() + 6 * kNum_mosqs, //[5]
                dxdt.begin() + 7 * kNum_mosqs, //[6]
                dxdt.begin() + 8 * kNum_mosqs, //[7]
                dxdt.begin() + 9 * kNum_mosqs //[8]
            ) )
        ) ),
        Fx(
            this->current_larva_to_immature,
            this->current_infected_to_infectious,
            this->current_eggs_to_larva
        )
    );
}

MosquitoManagerODEThrust::MosquitoManagerODEThrust(
        std::vector<float> max_larval_capacity_list
    ) {
    this->mosq_sys = new MosquitoEckhoff_multi(max_larval_capacity_list);

}

MosquitoManagerODEThrust::~MosquitoManagerODEThrust() {
    delete this->mosq_sys;
}

void MosquitoManagerODEThrust::step(
        const std::vector<float>& infection_prob_list
    ) {

    this->mosq_sys->pre_integration_update(infection_prob_list);

    odeint::integrate_const(
        this->ode_stepper,
        *(this->mosq_sys),
        this->mosq_sys->get_population(),
        0.0, 1.0, 0.01
    );
}

}